#include <iostream>
#include <stdio.h> 


#define ANSI_COLOR_RESET "\x1b[0m"
#define ANSI_COLOR_CYAN  "\x1B[36m"

int main() {  
  int nDevices;

  hipError_t err = hipGetDeviceCount(&nDevices);
  printf("%s\n", hipGetErrorString(err));
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

  for (int devIdx = 0; devIdx < nDevices; devIdx++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devIdx);
    printf("Device Number: %d\n", devIdx);
 