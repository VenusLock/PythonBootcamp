

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add_single_threaded(int n, float *x, float *y){
    for (int i = 0; i < n; i ++)
        y[i] = x[i] + y[i];
}
