

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add_single_threaded(int n, float *x, float *y){
    for (int i = 0; i < n; i ++)
        y[i] = x[i] + y[i];
}

__global__
void add_multi_threaded(int n, float *x, float *y){

    int idx = threadIdx.x;
    int stride = blockDim.x;
    // printf("threadIdx = %d; blockDim = %d; blockIdx = %d %d\n", idx, stride, blockIdx.x, blockIdx.y);
    for (int i = idx; i < n; i += stride)
    {
        printf("array idx %d\n", i);
        y[i] = x[i] + y[i];
    }
}

__global__
void add(int n, float *x, float *y)
{   
    // gird-stride loop.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // printf("threadIdx = %d; blockDim = %d; blockIdx = %d\n", idx, stride, blockIdx.x);
    for (int i = idx; i < n; i += stride)
        y[i] = x[i] + y[i];
}