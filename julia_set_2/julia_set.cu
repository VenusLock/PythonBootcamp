
// #include "../cuda_by_example/common/book.h"
#include "../cuda_by_example/common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    /* Struct for representing complex numbers */
    float real;
    float imag;

    __device__
    hipComplex( float a, float b): real(a), imag(b) {}

    __device__
    float magnitude2(void) {
        return real*real + imag*imag;
    }

    __device__
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(real * a.real - imag * a.imag, imag * a.real + real * a.imag);
    }

    __device__
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(real + a.real, imag + a.imag);
    }

};

__device__
int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)