
// #include "../cuda_by_example/common/book.h"
#include "../cuda_by_example/common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    /* Struct for representing complex numbers */
    float real;
    float imag;

    __device__
    hipComplex( float a, float b): real(a), imag(b) {}

    __device__
    float magnitude2(void) {
        return real*real + imag*imag;