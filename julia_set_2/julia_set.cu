
// #include "../cuda_by_example/common/book.h"
#include "../cuda_by_example/common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    /* Struct for representing complex numbers */
    float real;
    float imag;

    __device__