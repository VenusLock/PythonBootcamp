
// #include "../cuda_by_example/common/book.h"
#include "../cuda_by_example/common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex
{
    /* Struct for representing complex numbers */
    float real;
    float imag;

    __device__
    hipComplex( float a, float b): real(a), imag(b) {}

    __device__
    float magnitude2(void) {
        return real*real + imag*imag;
    }

    __device__
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(real * a.real - imag * a.imag, imag * a.real + real * a.imag);
    }

    __device__
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(real + a.real, imag + a.imag);
    }

};

__device__
int julia(int x, int y) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);