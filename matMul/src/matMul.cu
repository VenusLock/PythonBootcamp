#include "hip/hip_runtime.h"

/*
 ============================================================================
 Name        : matMul.cu
 Author      : taras-sereda
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

typedef struct {

	int width;
	int height;
	float *elements;
} Matrix;

__global__ void matMulKernel(Matrix A, Matrix B, Matrix C) {
