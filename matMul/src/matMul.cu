#include "hip/hip_runtime.h"

/*
 ============================================================================
 Name        : matMul.cu
 Author      : taras-sereda
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

typedef struct {

	int width;
	int height;
	float *elements;
} Matrix;

__global__ void matMulKernel(Matrix A, Matrix B, Matrix C) {

	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	float Cval = 0;

	if (row < A.height && col < B.width) {
		for (int e=0; e<A.width; e++){
			Cval += A.elements[A.width * row + e] *
					B.elements[B.width * e + col];
		}
		C.elements[C.width * row + col] = Cval;
	}
}

void init_matrix(float *elements, int width, int height) {