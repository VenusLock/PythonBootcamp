#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float* x, float* y) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) y[i] = a * x[i] + y[i];
} 


int main(int argc, char** argv) {

        int nDevices; 
        hipGetDeviceCount(&nDevices);  

        printf("Number of GPU devices %d\n", nDevices);

        for (int i = 0; i < nDevices; i++) {
       
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
           