#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float* x, float* y) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) y[i] = a * x[i] + y[i];
} 


int main(int argc, char** argv) {